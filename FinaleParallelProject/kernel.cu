#include "hip/hip_runtime.h"

#include"MyMacro.h"
#include"MyProto.h"
#include <stdio.h>

__device__ double caculateValueOfWeight(double parameter, int sign, double alpha)
{
	return (parameter*sign*alpha);
}

__global__ void updateWeights(double* weights, double* parameters,double* otherp, int sign, double alpha)
{
	int index = threadIdx.x;
	double value = weights[index];
	weights[index] = value + caculateValueOfWeight( parameters[index], sign, alpha);
	
}

hipError_t updateWeightsWithCuda(double * weights, double * parameters, double * alpha, int * sign, int dimensionSize)
{
	hipError_t cudaStatus;

	double* weightsOnGPU;
	double* parametersOnGPU;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return hipErrorUnknown;
	}

	cudaStatus = hipMalloc((void**)&weightsOnGPU, (dimensionSize + 1) * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc of weights failed");
		return hipErrorUnknown;
	}
	cudaStatus = hipMalloc((void**)&parametersOnGPU, (dimensionSize + 1) * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc of parameters failed");
		return hipErrorUnknown;
	}

	cudaStatus = hipMemcpy(weightsOnGPU, weights, (dimensionSize + 1) * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CudaMemCpy  of weights failed");
		return hipErrorUnknown;
	}

	cudaStatus = hipMemcpy(parametersOnGPU, parameters, (dimensionSize + 1) * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CudaMemCpy  of weights failed");
		return hipErrorUnknown;
	}

	updateWeights << <1, dimensionSize + 1 >> > (weightsOnGPU, parametersOnGPU,parameters, *sign, *alpha);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Update  of weights failed");
		return hipErrorUnknown;
	}

	cudaStatus = hipMemcpy(weights, weightsOnGPU, (dimensionSize + 1)*sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CudaMemCpy  of weights back failed");
		return hipErrorUnknown;
	}

	
	return cudaStatus;
}
